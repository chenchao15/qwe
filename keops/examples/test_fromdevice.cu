#include "hip/hip_runtime.h"
// test convolution
// compile with
//		nvcc -I.. -DCUDA_BLOCK_SIZE=192 -DMAXTHREADSPERBLOCK0=1024 -DSHAREDMEMPERBLOCK0=49152 -Wno-deprecated-gpu-targets -std=c++14 --use_fast_math -O3 -o test_fromdevice test_fromdevice.cu

// testing "from device" convolution, i.e. convolution which is performed on the device
// directly from device data

#include <algorithm>
#include <thrust/device_vector.h>
#include <stdio.h>

#include <keops_includes.h>

#include <chrono>
typedef std::chrono::high_resolution_clock Clock;

#define DIMPOINT 3
#define DIMVECT 1

__TYPE__ floatrand() {
    return ((__TYPE__) std::rand())/RAND_MAX-.5;    // random value between -.5 and .5
}

template < class V > void fillrandom(V& v) {
    generate(v.begin(), v.end(), floatrand);    // fills vector with random values
}

void DispValues(__TYPE__ *x, int N, int dim) {
  std::cout << std::endl;
  int k = 0;
  for(int i=0; i<N; i++) {
    for(int d=0; d<dim; d++) {
      std::cout << x[k] << " ";
      k++;
    }
    std::cout << std::endl;
  }
  for(int d=0; d<dim; d++)
    std::cout << "... ";
  std::cout << std::endl << std::endl;
}

using namespace keops;

int main(int argc, char **argv) {

    int deviceID = 0;
    hipSetDevice(deviceID);

    // symbolic expression of the function : a gaussian kernel
    auto x = Vi(0,DIMPOINT);
    auto y = Vj(1,DIMPOINT);
    auto beta = Vj(2,DIMVECT);
    
    auto f = Exp(-SqNorm2(x-y)) * beta; 

    std::cout << std::endl << "Function f : " << std::endl;
    std::cout << PrintFormula(f);
    std::cout << std::endl << std::endl;

    auto Sum_f = Sum_Reduction(f,0);

    // now we test ------------------------------------------------------------------------------

    int Nx;
    sscanf(argv[1], "%d", &Nx);
    
    std::vector<__TYPE__> vx(Nx*x.DIM);    fillrandom(vx); __TYPE__ *px = vx.data();
    thrust::device_vector<__TYPE__> vx_d(vx);
    __TYPE__ *x_d = thrust::raw_pointer_cast(vx_d.data());

    std::vector<__TYPE__> vy(Nx*DIMPOINT);    fillrandom(vy); __TYPE__ *py = vy.data();
    thrust::device_vector<__TYPE__> vy_d(vy);
    __TYPE__ *y_d = thrust::raw_pointer_cast(vy_d.data());
   
    std::vector<__TYPE__> vb(Nx*DIMVECT);     fillrandom(vb); __TYPE__ *pb = vb.data();
    thrust::device_vector<__TYPE__> vb_d(vb);
    __TYPE__ *b_d = thrust::raw_pointer_cast(vb_d.data());
   
    thrust::device_vector<__TYPE__> vres_d(Nx*Sum_f.DIM);
    __TYPE__ *res_d = thrust::raw_pointer_cast(vres_d.data());
    



    clock_t begin, end;

    std::cout << "blank run 1" << std::endl;
    begin = clock();
    EvalRed<GpuConv1D_FromDevice>(Sum_f,Nx, Nx, res_d, x_d, y_d, b_d);
    end = clock();
    std::cout << "time for blank run 1 : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;

    std::cout << "blank run 2" << std::endl;
    begin = clock();
    EvalRed<GpuConv1D_FromDevice>(Sum_f,Nx, Nx, res_d, x_d, y_d, b_d);
    end = clock();
    std::cout << "time for blank run 2 : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;




    int Ntest = 10;

    std::cout << "testing From_Device mode" << std::endl;

    //begin = clock();
    auto start = Clock::now();

    for(int i=0; i<Ntest; i++)
        EvalRed<GpuConv1D_FromDevice>(Sum_f,Nx, Nx, res_d, x_d, y_d, b_d);

    //end = clock();
    //std::cout << "time for "<< Ntest <<" GPU computations (1D scheme) : " << double(end - begin) / CLOCKS_PER_SEC << std::endl;
    auto stop = Clock::now();
    std::cout << "time = " 
                  << Ntest << "x "
                  << (float) std::chrono::duration_cast<std::chrono::nanoseconds>(stop - start).count() / (float) (1000000 * Ntest)
                  << " milliseconds" << std::endl;

    std::vector<__TYPE__> resgpu1D(Nx*Sum_f.DIM);
    hipMemcpy(resgpu1D.data(), res_d, Nx*Sum_f.DIM*sizeof(__TYPE__), hipMemcpyDeviceToHost);





}



